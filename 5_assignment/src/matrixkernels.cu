#include "hip/hip_runtime.h"
#include <iostream>
#include "matrixkernels.cuh"
#include "hip/hip_runtime_api.h"

extern "C"
int iDivUp( int a, int b ){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}


extern "C"
unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}


// vector operation: x = fac0*a op fac1*b
__global__ void
_gpu_vector_op_( int op, float fac0, float fac1, float *a, float *b, float *x, int dim )
{
	/* TASK 1.1b: implement the elementwise vector operations
	
	 	x = fac0 * a (op) fac1 * b
	
		with op = {+,-,*, NONE}.
		NONE means x = fac0 * a   (b might be NULL)

		
		HINT: remember to safeguard the index (the thread id might be larger than the array size)! 
		-> if the thread index is >= dim return!
		
	*/
}




// matrix vector multiplication: x = A*b op c
__global__ void
_gpu_matrix_vector_( int op, float *A, float *b, float *c, float *x, int dim )
{
	/* TASK 1.2b: implement the matrix vector multiplication
	
		x = A * b (op) c
	
		with op = {+,-,*,NONE}.
		NONE means x = A * b     (c might be NULL)

		HINT: remember to safeguard the index (the thread id might be larger than the array size)!
		-> if the thread index is >= dim return!
	*/
	
	
}




// returns SUM[d_a * d_b]
float gpuReduceSUM( float* d_a, float *d_b, float* d_x, int dim, int nBlocks, int nThreads ){

	/* TASK 1.3b: implement the vector multiplication and sum reduction

		d_x = d_a * d_b (element wise product)
		returns SUM[d_x]
		
		implement reduction as discussed in the lecture using shared memory.
		
	*/
	
	float sum = 0;
	return sum;
}

// x = A*a
extern "C" 
void multiplyMatrixVector( float *h_A, float *h_a, float *h_x, int dim )
{
	float *d_A, *d_a, *d_x;

	checkCudaErrors( hipMalloc( (void**) &d_A, dim * dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_a, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_x, dim * sizeof( float ) ) );

	checkCudaErrors( hipMemcpy( d_A, h_A, dim * dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_a, h_a, dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_x, h_x, dim * sizeof( float ), hipMemcpyHostToDevice ) );

	checkCudaErrors( hipDeviceSynchronize() );

	// x = A*a
	int nThreads = 128;
	int nBlocks = iDivUp( dim, nThreads );
	_gpu_matrix_vector_<<< nBlocks, nThreads, 0 >>>( NONE, d_A, d_a, NULL, d_x, dim );
	checkCudaErrors( hipDeviceSynchronize() );

	// copy solution from device to host
	checkCudaErrors( hipMemcpy( h_x, d_x, dim * sizeof( float ), hipMemcpyDeviceToHost ) );

	// release device memory
	checkCudaErrors( hipFree( d_A ) );
	checkCudaErrors( hipFree( d_a ) );
	checkCudaErrors( hipFree( d_x ) );
	
	
}


extern "C" 
void computeConjugateGradientGPU( float *h_A, float *h_b, float *h_x, int dim, float errorTolerance )
{
	int nThreads = 128;							// set the number of threads per block to use by default
	int nBlocks = iDivUp( dim, nThreads );
	
	float *d_A, *d_b, *d_x, *d_r, *d_p, *d_q, *d_tmp;
	float alpha, beta, rho = 0;

	//allocate device memory
	checkCudaErrors( hipMalloc( (void**) &d_A, dim * dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_b, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_x, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_r, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_p, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_q, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_tmp, dim * sizeof( float ) ) );
	
	// copy host to device
	checkCudaErrors( hipMemcpy( d_A, h_A, dim * dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_b, h_b, dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_x, h_x, dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipDeviceSynchronize() );

	// init CG
	// ALGORITHM: r_0 = b-Ax_0
	// r_0 = Ax_0 - b
	_gpu_matrix_vector_<<< nBlocks, nThreads, 0 >>>( CL_SUB, d_A, d_x, d_b, d_r, dim );
	checkCudaErrors( hipDeviceSynchronize() );
	

	// r_0 = -r_0
	_gpu_vector_op_<<< nBlocks, nThreads >>>( NONE, -1.0f, 0.0f, d_r, NULL, d_r, dim );
	checkCudaErrors( hipDeviceSynchronize() );
	
	// p_0 = r_0
	_gpu_vector_op_<<< nBlocks, nThreads >>>( NONE,  1.0f, 0.0f, d_r, NULL, d_p, dim );
	checkCudaErrors( hipDeviceSynchronize() );

	// CG needs max dim iterations
	int i = 0;
	float minRho = 1000000000;
	for( i = 0; i < dim; i++ ){	
		
		// rho_k = sum(r_k * r_k)
		rho = gpuReduceSUM(d_r, d_r, d_tmp, dim, nBlocks, nThreads );
		checkCudaErrors( hipDeviceSynchronize() );
		
		if (minRho > rho) {
			minRho = rho;
		}
		
		//printf("iteration #%d, with rho = %f", i, rho);
		std::cout << "iteration #" << i << ", with rho = " << rho << "          " << '\r' << std::flush;
		// check here for criterion
		if( rho < errorTolerance) {
			break;
		}
		
		// q_k = A*p_k
		_gpu_matrix_vector_<<< nBlocks, nThreads, 0 >>>( NONE, d_A, d_p, NULL, d_q, dim );
		checkCudaErrors( hipDeviceSynchronize() );
		
		// alpha_k = rho_k / sum(p_k * q_k)
		alpha = rho / gpuReduceSUM(d_p, d_q, d_tmp, dim, nBlocks, nThreads );
		checkCudaErrors( hipDeviceSynchronize() );
		
		 // x_(k+1) = x_k + alpha_k * p_k
		_gpu_vector_op_<<< nBlocks, nThreads >>>( CL_ADD, 1.0f, alpha, d_x, d_p, d_x, dim );
		checkCudaErrors( hipDeviceSynchronize() );
		
		// r_(k+1) = r_k + (-alpha_k * q_k)
		_gpu_vector_op_<<< nBlocks, nThreads >>>( CL_ADD, 1.0f, -alpha, d_r, d_q, d_r, dim );
		checkCudaErrors( hipDeviceSynchronize() );

		// beta_k = sum(r_(k+1) * r_(k+1)) / rho_k
		beta = gpuReduceSUM(d_r, d_r, d_tmp, dim, nBlocks, nThreads ) / rho;
		checkCudaErrors( hipDeviceSynchronize() );
		
		// p_(k+1) = r_(k+1) + beta_k * p_k
		_gpu_vector_op_<<< nBlocks, nThreads >>>( CL_ADD, 1.0f, beta, d_r, d_p, d_p, dim );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	rho = gpuReduceSUM(d_r, d_r, d_tmp, dim, nBlocks, nThreads );

	printf("\nSolution found at iteration #%d, with rho = %f\n", i, rho);
	printf("\nminrho was %f\n", minRho);
	
	// copy solution from device to host
	checkCudaErrors( hipMemcpy( h_x, d_x, dim * sizeof( float ), hipMemcpyDeviceToHost ) );

	// release device memory
	checkCudaErrors( hipFree( d_A ) );
	checkCudaErrors( hipFree( d_b ) );
	checkCudaErrors( hipFree( d_x ) );
	checkCudaErrors( hipFree( d_r ) );
	checkCudaErrors( hipFree( d_p ) );
	checkCudaErrors( hipFree( d_q ) );
	checkCudaErrors( hipFree( d_tmp ) );
}
